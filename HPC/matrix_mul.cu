#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#define size 10

__global__ void MatMul(int *a, int *b, int*c, int n)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int result = 0;
      for( int j=0;j<size; j++)
      {
          int p = *(a + ty*size + j);
          int q = *(b + j*size + tx);
          result =  result + p*q;
      }
       c[ty*size + tx] = result;
}



int main()
{
    int *A,*B,*C;
    A = (int*)malloc(size * size * sizeof(int));
    B = (int*)malloc(size * size * sizeof(int));
    C = (int*)malloc(size * size * sizeof(int));
    
    
    for(int i=0; i<size;i++)
    {
        for(int j=0; j<size; j++)
        {
            *(A + i*size + j) = rand()%10;
            *(B + i*size + j) = rand()%10;
        }
    }
    
    
    int *AD, *BD, *CD;
    
    hipMalloc(&AD, size*size*sizeof(int));
    hipMalloc(&BD, size*size*sizeof(int));
    hipMalloc(&CD, size*size*sizeof(int));
    
    hipMemcpy(AD, A, size*size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(BD, B, size*size*sizeof(int), hipMemcpyHostToDevice);
    
    MatMul<<<1,size*size>>>(AD, BD, CD, size);
    
    hipMemcpy(C, CD, size*size*sizeof(int), hipMemcpyDeviceToHost);
    
    
    
   printf("Matrix A: \n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%d ", *(A + i*size + j));
		}
		printf("\n");
	}
	printf("\n");
	printf("Matrix B: \n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%d ", *(B + i*size + j));
		}
		printf("\n");
	}
	printf("Product: \n");
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%d ", *(C + i*size + j));
		}
		printf("\n");
	}
	printf("\n");
    
    
    hipFree(AD);
    hipFree(BD);
    hipFree(CD);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}
