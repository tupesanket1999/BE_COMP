
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <time.h>
using namespace std;

__global__ void matrixVectorMultiplication(int *a, int *b, int *c, int n) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;

    if (row < n)
        for (int j = 0; j < n; j++) {
            sum = sum + a[row * n + j] * b[j];
        }

    c[row] = sum;
}
int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 32;

    a = new int[n * n];
    b = new int[n];
    c = new int[n];
    int *d = new int[n];
    int size = n * sizeof(int);
    hipMalloc(&a_dev, size * size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i * n + j] = i * n + j + 1; // rand()%n;
        }

        b[i] = i + 1; // rand()%n;
        // cout<<a[i]<<" ";
        // d[i]=a[i]+b[i];
    }

    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipMemcpy(a_dev, a, size * size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);

    if (n * n > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil((double)n / (double)threadsPerBlock.x);
        blocksPerGrid.y = ceil((double)n / (double)threadsPerBlock.y);
    }

    hipEventRecord(start);
    matrixVectorMultiplication<<<blocksPerGrid, threadsPerBlock>>>(a_dev, b_dev,
                                                                   c_dev, n);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time = 0.0;
    hipEventElapsedTime(&time, start, end);

    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);
    cout << "\nGPU Time Elapsed:  " << time;

    // CPU matrixVector multiplication
    clock_t t = clock();
    int sum = 0;
    for (int row = 0; row < n; row++) {
        sum = 0;
        for (int col = 0; col < n; col++) {
            sum = sum + a[row * n + col] * b[col];
        }
        d[row] = sum;
    }
    t = clock() - t;
    cout << "\nCPU Time Elapsed:  "
         << ((double)t); //((double)t)/CLOCKS_PER_SEC;

    int error = 0;
    for (int i = 0; i < n; i++) {
        error += d[i] - c[i];
        // cout<<" gpu "<<c[i]<<" CPU "<<d[i]<<endl;
    }

    cout << "Error : " << error;

    return 0;
}
